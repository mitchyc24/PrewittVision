#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime.h>
#include <stdio.h>



__global__ void apply_prewitt(unsigned char* grayscale_image, unsigned char* output_image, unsigned int width, unsigned int height) {
    //TODO: implement apply_prewitt 
}



extern "C" void applyPrewitt(unsigned char* host_grayscale_image, unsigned char* host_output_image, unsigned int width, unsigned int height) {
    const int imageSize = width * height * sizeof(unsigned char);
    const int blockSize = 16;
    const int gridSize = (imageSize + blockSize - 1) / blockSize;

    unsigned char* device_grayscale_image;
    unsigned char* device_output_image;

    // CUDA memory operations
    hipError_t err;
    err = hipMalloc((void**)&device_grayscale_image, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device_grayscale_image: %s\n", hipGetErrorString(err));
    }

    err = hipMalloc((void**)&device_output_image, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device_output_image: %s\n", hipGetErrorString(err));
    }

    err = hipMemcpy(device_grayscale_image, host_grayscale_image, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying to device_grayscale_image: %s\n", hipGetErrorString(err));
    }

    apply_prewitt<<<gridSize, blockSize>>>(device_grayscale_image, device_output_image, width, height);
    hipDeviceSynchronize();

    err = hipMemcpy(host_output_image, device_output_image, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying to host_output_image: %s\n", hipGetErrorString(err));
    }

    hipFree(device_grayscale_image);
    hipFree(device_output_image);
}


