#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime.h>


__global__ void apply_prewitt(unsigned char* grayscale_image, unsigned char* output_image) {
    //TODO: implement apply_prewitt 
}



void applyPrewitt(unsigned char* host_grayscale_image, unsigned char* host_output_image) {
    const int imageSize = 1024 * 1024;
    const int blockSize = 16;
    const int gridSize = imageSize / blockSize;

    unsigned char* device_grayscale_image;
    unsigned char* device_output_image;

    // CUDA memory operations
    hipMalloc((void**)&device_grayscale_image, imageSize);
    hipMalloc((void**)&device_output_image, imageSize);

    hipMemcpy(device_grayscale_image, host_grayscale_image, imageSize, hipMemcpyHostToDevice);

    apply_prewitt<<<gridSize, blockSize>>>(device_grayscale_image, device_output_image);
    hipDeviceSynchronize();

    hipMemcpy(host_output_image, device_output_image, imageSize, hipMemcpyDeviceToHost);

    hipFree(device_grayscale_image);
    hipFree(device_output_image);
}
