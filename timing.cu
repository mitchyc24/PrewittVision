#include "timing.h"
#include <stdio.h>
#include <functional>

float timeKernelExecution(const std::function<void()>& kernelFunction) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    kernelFunction(); // Execute the kernel function
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedMilliseconds = 0;
    hipEventElapsedTime(&elapsedMilliseconds, start, stop);

    printf("Kernel execution time: %.3f ms\n", elapsedMilliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedMilliseconds; // Return the elapsed time in milliseconds
}
