#include "timing.h"
#include <stdio.h>

void timeKernelExecution(const std::function<void()>& kernelFunction) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    kernelFunction(); // Execute the kernel function
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedMilliseconds = 0;
    hipEventElapsedTime(&elapsedMilliseconds, start, stop);

    printf("Kernel execution time: %.3f ms\n", elapsedMilliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
