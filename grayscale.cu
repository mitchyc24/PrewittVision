#include "hip/hip_runtime.h"
#include "kernels.h"
#include "timing.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void convert_to_grayscale(unsigned char* input_image, unsigned char* grayscale_image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < width && y < height) {
        int pixelIndex = y * width + x;
        unsigned char r = input_image[4 * pixelIndex];
        unsigned char g = input_image[4 * pixelIndex + 1];
        unsigned char b = input_image[4 * pixelIndex + 2];

        // Calculate the grayscale value
        grayscale_image[pixelIndex] = (unsigned char)(0.299f * r + 0.587f * g + 0.114f * b);
    }
}

extern "C" float convertToGrayscale(unsigned char* host_input_image, unsigned char* host_grayscale_image, unsigned int width, unsigned int height) {
    const int imageSize = width * height * sizeof(unsigned char);
    const int blockSize = 16;
    dim3 blockDims(blockSize, blockSize, 1);
    dim3 gridDims((width + blockDims.x - 1) / blockDims.x, (height + blockDims.y - 1) / blockDims.y, 1);

    unsigned char* device_input_image;
    unsigned char* device_grayscale_image;

    // CUDA memory operations
    hipError_t err;
    err = hipMalloc((void**)&device_input_image, imageSize * 4); // For RGBA
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device_input_image: %s\n", hipGetErrorString(err));
    }

    err = hipMalloc((void**)&device_grayscale_image, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device_grayscale_image: %s\n", hipGetErrorString(err));
    }

    err = hipMemcpy(device_input_image, host_input_image, imageSize * 4, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying to device_input_image: %s\n", hipGetErrorString(err));
    }

    auto kernelFunction = [&]() {
        convert_to_grayscale<<<gridDims, blockDims>>>(device_input_image, device_grayscale_image, width, height);
    };

    // Call and time the kernel execution using the timing function
    printf("Launching Grayscale Kernel\n");
    float elapsedTime = timeKernelExecution(kernelFunction);


    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error launching kernel: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    err = hipMemcpy(host_grayscale_image, device_grayscale_image, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying to host_grayscale_image: %s\n", hipGetErrorString(err));
    }

    hipFree(device_input_image);
    hipFree(device_grayscale_image);

    return elapsedTime;
}
