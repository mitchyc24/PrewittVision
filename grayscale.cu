#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime.h>
#include <stdio.h>



__global__ void convert_to_grayscale(unsigned char* input_image, unsigned char* grayscale_image, unsigned int width, unsigned int height) {
    //TODO: implement grayscale conversion
}


extern "C" void convertToGrayscale(unsigned char* host_input_image, unsigned char* host_grayscale_image, unsigned int width, unsigned int height) {
    const int imageSize = width * height * sizeof(unsigned char);
    const int blockSize = 16;
    const int gridSize = (imageSize + blockSize - 1) / blockSize;

    unsigned char* device_input_image;
    unsigned char* device_grayscale_image;

    // CUDA memory operations
    hipError_t err;
    err = hipMalloc((void**)&device_input_image, imageSize * 4); // For RGBA
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device_input_image: %s\n", hipGetErrorString(err));
    }

    err = hipMalloc((void**)&device_grayscale_image, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device_grayscale_image: %s\n", hipGetErrorString(err));
    }

    err = hipMemcpy(device_input_image, host_input_image, imageSize * 4, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying to device_input_image: %s\n", hipGetErrorString(err));
    }

    convert_to_grayscale<<<gridSize, blockSize>>>(device_input_image, device_grayscale_image, width, height); // Assuming kernel takes width and height
    hipDeviceSynchronize();

    err = hipMemcpy(host_grayscale_image, device_grayscale_image, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying to host_grayscale_image: %s\n", hipGetErrorString(err));
    }

    hipFree(device_input_image);
    hipFree(device_grayscale_image);
}
